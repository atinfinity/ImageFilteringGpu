#include "hip/hip_runtime.h"
﻿#include "imageFilteringGpu.cuh"

#include <opencv2/core/cuda/common.hpp>
#include <opencv2/cudev.hpp>
#include <hip/hip_runtime.h>
#include <>

texture<uchar, hipTextureType2D, hipReadModeElementType> srcTex(false, hipFilterModePoint, hipAddressModeClamp);

__device__ uchar clipGpu(float val)
{
	return (val < 0.0f) ? 0 : (val > 255.0f) ? 255 : (uchar)val;
}

__global__ void imageFilteringGpu
(
    const cv::cudev::PtrStepSz<uchar> src,
    cv::cudev::PtrStepSz<uchar> dst,
    const cv::cudev::PtrStepSz<float> kernel, 
    const int border_size
)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if((y >= border_size) && y < (dst.rows-border_size)){
        if((x >= border_size) && (x < (dst.cols-border_size))){
            float sum = 0.0f;
            for(int yy = 0; yy < kernel.rows; yy++){
                for(int xx = 0; xx < kernel.cols; xx++){
                    sum = __fadd_rn(sum, __fmul_rn(kernel.ptr(yy)[xx], src.ptr(y+yy-border_size)[x+xx-border_size]));
                }
            }
            dst.ptr(y)[x] = clipGpu(sum);
        }
    }
}

// use __ldg
__global__ void imageFilteringGpu_ldg
(
    const cv::cudev::PtrStepSz<uchar> src,
    cv::cudev::PtrStepSz<uchar> dst,
    const cv::cudev::PtrStepSz<float> kernel, 
    const int border_size
)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if((y >= border_size) && y < (dst.rows-border_size)){
        if((x >= border_size) && (x < (dst.cols-border_size))){
            float sum = 0.0f;
            for(int yy = 0; yy < kernel.rows; yy++){
                const uchar* psrc = src.ptr(y+yy-border_size) + (x-border_size);
                const float* pkernel = kernel.ptr(yy);
                for(int xx = 0; xx < kernel.cols; xx++){
                    sum = __fadd_rn(sum, __fmul_rn(__ldg(&pkernel[xx]), __ldg(&psrc[xx])));
                }
            }
            dst.ptr(y)[x] = sum;
        }
    }
}

// use texture
__global__ void imageFilteringGpu_tex
(
    const cv::cudev::PtrStepSz<uchar> src,
    cv::cudev::PtrStepSz<uchar> dst,
    const cv::cudev::PtrStepSz<float> kernel, 
    const int border_size
)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if((y >= border_size) && (y < (dst.rows-border_size))){
        if((x >= border_size) && (x < (dst.cols-border_size))){
            float sum = 0.0f;
            for(int yy = 0; yy < kernel.rows; yy++){
                for(int xx = 0; xx < kernel.cols; xx++){
                    sum = __fadd_rn(sum, __fmul_rn(kernel.ptr(yy)[xx], tex2D(srcTex, x + xx - border_size, y + yy - border_size)));
                }
            }
            dst.ptr(y)[x] = sum;
        }
    }
}

void launchImageFilteringGpu
(
    cv::cuda::GpuMat& src,
    cv::cuda::GpuMat& dst,
    cv::cuda::GpuMat& kernel, 
    const int border_size
)
{
    cv::cudev::PtrStepSz<uchar> pSrc =
        cv::cudev::PtrStepSz<uchar>(src.rows, src.cols * src.channels(), src.ptr<uchar>(), src.step);

    cv::cudev::PtrStepSz<uchar> pDst =
        cv::cudev::PtrStepSz<uchar>(dst.rows, dst.cols * dst.channels(), dst.ptr<uchar>(), dst.step);

    cv::cudev::PtrStepSz<float> pKernel =
        cv::cudev::PtrStepSz<float>(kernel.rows, kernel.cols * kernel.channels(), kernel.ptr<float>(), kernel.step);

    const dim3 block(64, 2);
    const dim3 grid(cv::cudev::divUp(dst.cols, block.x), cv::cudev::divUp(dst.rows, block.y));

    imageFilteringGpu<<<grid, block>>>(pSrc, pDst, pKernel, border_size);

    CV_CUDEV_SAFE_CALL(hipGetLastError());
    CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());
}

// use __ldg
void launchImageFilteringGpu_ldg
(
    cv::cuda::GpuMat& src,
    cv::cuda::GpuMat& dst,
    cv::cuda::GpuMat& kernel, 
    const int border_size
)
{
    cv::cudev::PtrStepSz<uchar> pSrc =
        cv::cudev::PtrStepSz<uchar>(src.rows, src.cols * src.channels(), src.ptr<uchar>(), src.step);

    cv::cudev::PtrStepSz<uchar> pDst =
        cv::cudev::PtrStepSz<uchar>(dst.rows, dst.cols * dst.channels(), dst.ptr<uchar>(), dst.step);

    cv::cudev::PtrStepSz<float> pKernel =
        cv::cudev::PtrStepSz<float>(kernel.rows, kernel.cols * kernel.channels(), kernel.ptr<float>(), kernel.step);

    const dim3 block(64, 2);
    const dim3 grid(cv::cudev::divUp(dst.cols, block.x), cv::cudev::divUp(dst.rows, block.y));

    imageFilteringGpu_ldg<<<grid, block>>>(pSrc, pDst, pKernel, border_size);

    CV_CUDEV_SAFE_CALL(hipGetLastError());
    CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());
}

// use texture
void launchImageFilteringGpu_tex
(
    cv::cuda::GpuMat& src,
    cv::cuda::GpuMat& dst,
    cv::cuda::GpuMat& kernel, 
    const int border_size
)
{
    cv::cudev::PtrStepSz<uchar> pSrc =
        cv::cudev::PtrStepSz<uchar>(src.rows, src.cols * src.channels(), src.ptr<uchar>(), src.step);

    cv::cudev::PtrStepSz<uchar> pDst =
        cv::cudev::PtrStepSz<uchar>(dst.rows, dst.cols * dst.channels(), dst.ptr<uchar>(), dst.step);

    cv::cudev::PtrStepSz<float> pKernel =
        cv::cudev::PtrStepSz<float>(kernel.rows, kernel.cols * kernel.channels(), kernel.ptr<float>(), kernel.step);

    // bind texture
    cv::cuda::device::bindTexture<uchar>(&srcTex, pSrc);

    const dim3 block(64, 2);
    const dim3 grid(cv::cudev::divUp(dst.cols, block.x), cv::cudev::divUp(dst.rows, block.y));

    imageFilteringGpu_tex<<<grid, block>>>(pSrc, pDst, pKernel, border_size);

    CV_CUDEV_SAFE_CALL(hipGetLastError());
    CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());

    // unbind texture
    CV_CUDEV_SAFE_CALL(hipUnbindTexture(srcTex));
}

double launchImageFilteringGpu
(
    cv::cuda::GpuMat& src,
    cv::cuda::GpuMat& dst,
    cv::cuda::GpuMat& kernel, 
    const int border_size, 
    const int loop_num
)
{
    double f = 1000.0f / cv::getTickFrequency();
    int64 start = 0, end = 0;
    double time = 0.0;
    for (int i = 0; i <= loop_num; i++){
        start = cv::getTickCount();
        launchImageFilteringGpu(src, dst, kernel, border_size);
        end = cv::getTickCount();
        time += (i > 0) ? ((end - start) * f) : 0;
    }
    time /= loop_num;

    return time;
}

double launchImageFilteringGpu_ldg
(
    cv::cuda::GpuMat& src,
    cv::cuda::GpuMat& dst,
    cv::cuda::GpuMat& kernel, 
    const int border_size, 
    const int loop_num
)
{
    double f = 1000.0f / cv::getTickFrequency();
    int64 start = 0, end = 0;
    double time = 0.0;
    for (int i = 0; i <= loop_num; i++){
        start = cv::getTickCount();
        launchImageFilteringGpu_ldg(src, dst, kernel, border_size);
        end = cv::getTickCount();
        time += (i > 0) ? ((end - start) * f) : 0;
    }
    time /= loop_num;

    return time;
}

double launchImageFilteringGpu_tex
(
    cv::cuda::GpuMat& src,
    cv::cuda::GpuMat& dst,
    cv::cuda::GpuMat& kernel, 
    const int border_size, 
    const int loop_num
)
{
    double f = 1000.0f / cv::getTickFrequency();
    int64 start = 0, end = 0;
    double time = 0.0;
    for (int i = 0; i <= loop_num; i++){
        start = cv::getTickCount();
        launchImageFilteringGpu_tex(src, dst, kernel, border_size);
        end = cv::getTickCount();
        time += (i > 0) ? ((end - start) * f) : 0;
    }
    time /= loop_num;

    return time;
}
